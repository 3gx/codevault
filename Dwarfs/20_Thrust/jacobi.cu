#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include "timer.h"


void jacobi_gold(
    thrust::host_vector<float> &output,
    const thrust::host_vector<float> &input, 
    const thrust::host_vector<float> &source, 
    float constant)
{
  const auto N = static_cast<int>(std::sqrt(input.size()));
  assert((int)input.size()== N*N);

  auto u = [&](int i, int j) -> float& { return output[j*N+i]; };
  auto g = [&](int i, int j) { return input[j*N+i]; };
  auto f = [&](int i, int j) { return source[j*N+i]; };


  //u(i,j) = 0.25*(g(i+1, j) + g(i-1, j) + g(i, j+1) + g(i, j-1) + constant * f(i,j));
  for (int j = 0; j < N; j++)
    for (int i = 0; i < N; i++)
    {
      auto im = i-1;
      auto ip = i+1;
      auto jm = j-1;
      auto jp = j+1;
      if (im <  0) im += N;
      if (ip >= N) ip -= N;
      if (jm <  0) jm += N;
      if (jp >= N) jp -= N;
      u(i,j) = 0.25f*(g(ip,j)+g(im,j)+g(i,jp)+g(i,jm) + constant*f(i,j));
    }
}

bool diff(const thrust::host_vector<float> &x, const thrust::host_vector<float> &y)
{
  using namespace thrust;
  const auto n = x.size();
  for (size_t i = 0; i < n; i++)
  {
    const auto z = x[i] - y[i];
    if (abs(z) > 1.0e-7)
      return false;
  }
  return true;
}

struct jacobi_functor
{
  int N;
  float constant;
  thrust::device_ptr<float> input,source;
  thrust::device_ptr<float> output;
  __host__ __device__
  void operator()(int idx)
  {
    using namespace thrust;
    auto u = [&](int i, int j) { return output[j*N+i]; };
    auto g = [&](int i, int j) { return  input[j*N+i]; };
    auto f = [&](int i, int j) { return source[j*N+i]; };
    
    auto j = idx/N;
    auto i = idx - j*N;
    auto im = i-1;
    auto ip = i+1;
    auto jm = j-1;
    auto jp = j+1;
    if (im <  0) im += N;
    if (ip >= N) ip -= N;
    if (jm <  0) jm += N;
    if (jp >= N) jp -= N;
    u(i,j) = 0.25f*(g(ip,j)+g(im,j)+g(i,jp)+g(i,jm) + constant*f(i,j));
  }
};



int main(void)
{
  using namespace thrust;

  const auto N = 2048;
  host_vector<float> input(N*N), source(N*N), result(N*N);

  for (int j = 0; j < N; j++)
    for (int i = 0; i < N; i++)
    {
       input[j*N+i] = (i+j)&2;
      source[j*N+i] = (i+j)&4;
    }
  const auto constant = 0.25f;


  jacobi_gold(result, input,source,constant);

  int nrep = 10;

  Timer timer_gold  ("gold  ");
  Timer timer_thrust("thrust");
  for (int rep = 0; rep < nrep; rep++)
  {
    timer_gold.tbeg();
    jacobi_gold(result, input,source,rep/8.0f+constant);
    timer_gold.tend();
  }


  device_vector<float> d_input  = input;
  device_vector<float> d_source = source;
  device_vector<float> d_result = input;

  using CountingIterator = typename thrust::counting_iterator<int>;

  for_each(CountingIterator(0), CountingIterator(N*N),
      jacobi_functor{N,constant,
      d_input.data(),
      d_source.data(),
      d_result.data()});
      hipDeviceSynchronize();
  for (int rep = 0; rep < nrep; rep++)
  {
    timer_thrust.tbeg();
    for_each(CountingIterator(0), CountingIterator(N*N),
        jacobi_functor{N,rep/8.0f + constant,
        d_input.data(),
        d_source.data(),
        d_result.data()});
    hipDeviceSynchronize();
    timer_thrust.tend();
  }

  
  assert(diff(result, d_result));

  std::cout << "OK\n";

  timer_gold.finalize();
  timer_thrust.finalize();
  auto bw = [&](double dt, double ddt) { std::cout << " - BW= " << N*N*sizeof(float)*3/dt/1e9 << " GB/s "; };
  timer_gold  .print(bw);
  timer_thrust.print(bw);
  std::cout << " speedup: " << timer_gold.dtmean()/timer_thrust.dtmean() << "x \n";

  return 0;
}
