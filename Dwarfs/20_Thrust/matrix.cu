#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>


using namespace std;


template<size_t... I>
struct indexSeq {};


template<size_t N, size_t... S>
struct makeIndexSeqImpl : makeIndexSeqImpl<N-1,N-1,S...> {};
template<size_t... S>
struct makeIndexSeqImpl<0,S...>
{
    using type = indexSeq<S...>;
};
template<size_t N>
using makeIndexSeq = typename makeIndexSeqImpl<N>::type;




template <int r, int c>
class mymatrix
{
  public:
    __host__ __device__
      mymatrix(){};
    __host__ __device__
      ~mymatrix(){};


    template<typename  Tuple  ,int which, size_t... I>
      __host__ __device__ 
      auto unpack(Tuple t,indexSeq<I...>) -> std::initializer_list<double>
      {
        // NO IDEA HOW TO GET AROUND THE HARD CODING HERE
        auto foo = {data[I]=thrust::get<I>(thrust::get<which>(t))...};
        return foo; /* removes compiler warning */
      }


    template<typename Tuple ,int which, size_t... I>
      __host__ __device__ 
      auto pack(Tuple t,indexSeq<I...>) -> std::initializer_list<double>
      {
        // NO IDEA HOW TO GET AROUND THE HARD CODING HERE
        auto foo = {thrust::get<I>(thrust::get<which>(t))=data[I]...};
        return foo; /* removes compiler warning */
      }
    
    double data[r*c];
};


template<int rowsleft,int colsleft,int rowsright,int colsright>
  __host__ __device__ 
void multiply_mat(mymatrix<rowsleft,colsleft>&m1,mymatrix<rowsleft,colsleft>&m2,mymatrix<rowsleft,colsright>&m3)
{
  for (int row=0;row<rowsleft;row++)
    for (int col=0;col<colsright;col++)
    {
      m3.data[col+row*colsright]=0.;
      for (int sum_i=0;sum_i<colsleft;sum_i++)
        m3.data[col+row*colsright]+=m1.data[sum_i+row*colsleft]*m2.data[col+sum_i*colsright]; //THIS IS NICE (PROBABLY SOME BLAS STUFF WOULD BE EVEN NICER, THATS NOT THE POINT THO)
    }
};


template<int r1,int c1,int r2,int c2>
struct my_matrix_functor
{
  mymatrix<r1,c1> m1;
  mymatrix<r2,c2> m2;
  mymatrix<r1,c2> m3;
  my_matrix_functor(){};
  template<typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
      static_assert(r1*c2 == r2*c2, "Matrices are not of the same size");
      m1.template unpack<Tuple,0>(t,makeIndexSeq<r1*c1>());
      m2.template unpack<Tuple,1>(t,makeIndexSeq<r1*c1>());
      multiply_mat<r1,c1,r2,c2>(m1,m2,m3);
      m3.template pack<Tuple,2>(t,makeIndexSeq<r1*c1>());  
    }
};


template<size_t... I, typename T, typename Func>
void my_foreach(const T& matrices1_SoA, const T& matrices2_SoA, T& matrices3_SoA, Func func, indexSeq<I...>)
{
  thrust::for_each(
      thrust::make_zip_iterator(
        thrust::make_tuple(
          thrust::make_zip_iterator(thrust::make_tuple(matrices1_SoA[I].begin()...)),
          thrust::make_zip_iterator(thrust::make_tuple(matrices2_SoA[I].begin()...)),
          thrust::make_zip_iterator(thrust::make_tuple(matrices3_SoA[I].begin()...))
          )
        ),
      thrust::make_zip_iterator(
        thrust::make_tuple(
          thrust::make_zip_iterator(thrust::make_tuple(matrices1_SoA[I].end()...)),
          thrust::make_zip_iterator(thrust::make_tuple(matrices2_SoA[I].end()...)),
          thrust::make_zip_iterator(thrust::make_tuple(matrices3_SoA[I].end()...))
          )
        )
      , func);
}


int  main()
{
  //do  A*B -> C for 10000 matrices
  // size(A) = M*M
  // size(B) = M*M
  // size(C) = M*M
  
  constexpr auto M = size_t{2};


  vector<thrust::device_vector<double> > matrices1_SoA(M*M, thrust::device_vector<double>(10000,1.)); //left matrices
  vector<thrust::device_vector<double> > matrices2_SoA(M*M, thrust::device_vector<double>(10000,2.)); //right matrices
  vector<thrust::device_vector<double> > matrices3_SoA(M*M, thrust::device_vector<double>(10000,0.)); //result matrices


  //NO IDEA HOW THE CALL COULD BE PARAMETRIZED
  // use variadic templates...
  my_foreach(matrices1_SoA, matrices2_SoA, matrices3_SoA,  my_matrix_functor<M,M,M,M>(), makeIndexSeq<M*M>());


  cout<<"OUTPUT MATRIX:"<<endl;
  cout<<matrices3_SoA[0][0]<<" "<<matrices3_SoA[1][0]<<endl;
  cout<<matrices3_SoA[2][0]<<" "<<matrices3_SoA[3][0]<<endl;


  return 1;
}
