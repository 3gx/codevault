#include "hip/hip_runtime.h"
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "timer.h"

typedef double value_type;

class varset {
public: 
	value_type v0, v1, v2, v3, v4, v5, v6, v7, v8, v9, v10;
	__host__ __device__ varset() : v0(0.0), v1(1.0), v2(1.5), v3(1.75), v4(1.875), v5(1.9375), v6(1.9688), v7(1.9922), v8(1.9961), v9(1.9980), v10(1.9990) { }
};

//conditionally compile for the GPU or the CPU
#ifdef GPU

typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< varset > var_type;

#else

typedef thrust::host_vector< value_type > state_type;
typedef thrust::host_vector< varset > var_type;

#endif

#if 0
#include <trove/block.h>

template<typename T, int s>
__global__ void test_block_copy(const T* x, T* r, int l) {
    typedef trove::array<T, s> s_ary;
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;

    for(int index = global_index; index < l; index += gridDim.x * blockDim.x) {

        //The block memory accesses only function
        //correctly if the warp is converged. Here we check.
        if (trove::warp_converged()) {
            //Warp converged, indices are contiguous, call the fast
            //load and store
            s_ary d = trove::load_array_warp_contiguous<s>(x, index);
            trove::store_array_warp_contiguous(r, index, d);
        } else {
            //Warp not converged, call the slow load and store
            s_ary d = trove::load_array<s>(x, index);
            trove::store_array(r, index, d);
        }
    }
}
#endif


struct updatevars_functor {

   	template< class Tuple >
	__host__ __device__
	void operator()( Tuple tuple_in ) {
		// dynamics at dv_n/dt = v_{n-1} - v_n
		varset var      = thrust::get<0>(tuple_in);
		value_type dt   = thrust::get<1>(tuple_in);
    	    	varset varnew;

		varnew.v0  = var.v0  + dt*( var.v10 - var.v0  );
		varnew.v1  = var.v1  + dt*( var.v0  - var.v1  );
		varnew.v2  = var.v2  + dt*( var.v1  - var.v2  );
		varnew.v3  = var.v3  + dt*( var.v2  - var.v3  );
		varnew.v4  = var.v4  + dt*( var.v3  - var.v4  );
		varnew.v5  = var.v5  + dt*( var.v4  - var.v5  );
		varnew.v6  = var.v6  + dt*( var.v5  - var.v6  );
		varnew.v7  = var.v7  + dt*( var.v6  - var.v7  );
		varnew.v8  = var.v8  + dt*( var.v7  - var.v8  );
		varnew.v9  = var.v9  + dt*( var.v8  - var.v9  );
		varnew.v10 = var.v10 + dt*( var.v9  - var.v10 );
		
		thrust::get<0>(tuple_in) = varnew;
	}
};

int main( int arc , char* argv[] )
{	
	if (arc!=2) exit(-1);
	int N = atoi(argv[1]); // number of copies to simulation

	// initial condition
	var_type var(N);
	
	// numerical parameters
	value_type t  = 0;
	value_type dt = 0.001;
	value_type tf = 100;

	//setup
	thrust::constant_iterator<value_type> dtval(dt);

	// solve the ode
  {
    Timer ts_("simulation", Timer::verbose_destructor{});
    ts_.tbeg();
    Timer t_("step",Timer::verbose_destructor{});
    while ( t < tf ) {
      // update the variables
      t_.tbeg();
      thrust::for_each_n(
          thrust::make_zip_iterator(thrust::make_tuple(var.begin(),dtval)),
          N,
          updatevars_functor()
          );
      hipDeviceSynchronize();
      t_.tend();
      t += dt;
    }
    ts_.tend();
  }

	
	return 0;
}
